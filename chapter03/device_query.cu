#include<stdio.h>

int main(){

    hipDeviceProp_t prop;

    int count = 0;

    hipGetDeviceCount(&count);

    printf("CUDA Capable Devices:\t%d\n", count);

    for(int i = 0; i < count ; i++){
        hipGetDeviceProperties(&prop, i);
        printf("General information for device %d\n", i);
        printf("Name:\t%s\n",prop.name);
        printf("Cupute Capability:\t%d.%d\n", prop.major,  prop.minor);
        printf("Clock Rate:\t%d\n", prop.clockRate);
        printf("Device copy overlap:\t");
            if(prop.deviceOverlap){
                printf("Enabled.\n");
            }
            else{
                printf("Disabled.\n");
            }
        printf("Kernel Execution Timeout:\t");
            if(prop.kernelExecTimeoutEnabled){
                printf("Enabled.\n");
            }
            else{
                printf("Disabled.\n");
            }
        printf("\n");

        printf("Memory information for device %d\n", i);
        printf("Total Global Memory:\t%ld\n", prop.totalGlobalMem);
        printf("Total Constant Memory:\t%ld\n", prop.totalConstMem);
        printf("Maximum Memory Pitch:\t%ld\n", prop.memPitch);
        printf("Texture Alignment:\t%ld\n", prop.textureAlignment);
        printf("\n");

        printf("Multi Processor Information for device %d\n", i);
        printf("Multiprocessor Count:\t%d\n", prop.multiProcessorCount);
        printf("Shared Memory for MP:\t%ld\n", prop.sharedMemPerBlock);
        printf("Registers for MP:\t%ld\n", prop.regsPerBlock);
        printf("Threads in warp:\t%d\n", prop.warpSize);
        printf("Maxmimum threads per block:\t%d\n", prop.maxThreadsPerBlock);
        printf("Maxmimum threads dimensions:\t(%d,%d,%d)\n",
                prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max Grid Dimensionss:\t(%d,%d,%d)\n",
                prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    }

    return 0;
}
