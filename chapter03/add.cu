
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__
void add(int a, int b, int *c){

    *c = a+b;

    return;
}

int main(){

    hipSetDevice(0);

    int c;
    int *dev_c;

    hipMalloc(&dev_c, sizeof(int));

    add<<<1,1>>>(2,7,dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("2+7=%d\n",c);
    hipFree(dev_c);

    return 0;
}
