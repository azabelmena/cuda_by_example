
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

const int SIZE = 100000;

__global__
void populate(int *a){

    int i = blockIdx.x;
    if(i<SIZE){
        a[i]=i;
    }

    return;
}

void print(int *a){
    for(int i = 0; i < SIZE ; i++){
        printf("%d\t",a[i]);
    }
    printf("\n");

    return;
}

__global__
void vector_add(int *a, int *b, int *c){

    int i = blockIdx.x;

    if(i < SIZE){
        c[i] = a[i]+b[i];
    }
}

int main(){

    int a[SIZE];
    int b[SIZE];
    int c[SIZE];

    int *dev_a;
    int *dev_b;
    int *dev_c;

    hipMalloc(&dev_a, sizeof(a));
    hipMalloc(&dev_b, sizeof(b));
    hipMalloc(&dev_c, sizeof(c));

    hipMemcpy(dev_a, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(b), hipMemcpyHostToDevice);

    populate<<<SIZE, 1>>>(dev_a);
    populate<<<SIZE, 1>>>(dev_b);

    vector_add<<<SIZE, 1>>>(dev_a,dev_b,dev_c);


    hipMemcpy(c, dev_c, sizeof(c), hipMemcpyDeviceToHost);

    print(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
